#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "material.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "mathUtil.h"
#include "sampler.h"

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* Image, int toneMapping) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);

		// Tonemapping and gamma correction
		glm::vec3 color = Image[index] / float(iter);

		switch (toneMapping) {
		case ToneMapping::Filmic:
			color = Math::filmic(color);
			break;
		case ToneMapping::ACES:
			color = Math::ACES(color);
			break;
		case ToneMapping::None:
			break;
		}
		color = Math::correctGamma(color);
		glm::ivec3 iColor = glm::clamp(glm::ivec3(color * 255.f), glm::ivec3(0), glm::ivec3(255));

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = iColor.x;
		pbo[index].y = iColor.y;
		pbo[index].z = iColor.z;
	}
}

#define PixelIdxForTerminated -1

static Scene* hstScene = nullptr;
static GuiDataContainer* guiData = nullptr;
static glm::vec3* devImage = nullptr;
static PathSegment* devPaths = nullptr;
static PathSegment* devTerminatedPaths = nullptr;
static Intersection* devIntersections = nullptr;
static int* devIntersecMatKeys = nullptr;
static int* devSegmentMatKeys = nullptr;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static thrust::device_ptr<PathSegment> devPathsThr;
static thrust::device_ptr<PathSegment> devTerminatedPathsThr;

static thrust::device_ptr<Intersection> devIntersectionsThr;
static thrust::device_ptr<int> devIntersecMatKeysThr;
static thrust::device_ptr<int> devSegmentMatKeysThr;

static glm::vec3* devGBufferPos = nullptr;
static glm::vec3* devGBufferNorm = nullptr;

#if ENABLE_GBUFFER
static Intersection* devGBuffer = nullptr;
#endif
 
void InitDataContainer(GuiDataContainer* imGuiData) {
	guiData = imGuiData;
}

#if ENABLE_GBUFFER
#endif

__global__ void renderGBuffer(DevScene* scene, Camera cam, Intersection *GBuffer) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	if (idx >= cam.resolution.x || idy >= cam.resolution.y) {
		return;
	}

	float aspect = float(cam.resolution.x) / cam.resolution.y;
	float tanFovY = glm::tan(glm::radians(cam.fov.y));
	glm::vec2 pixelSize = 1.f / glm::vec2(cam.resolution);
	glm::vec2 scr = glm::vec2(idx, idy) * pixelSize;
	glm::vec2 ruv = scr + pixelSize * glm::vec2(.5f);
	ruv = 1.f - ruv * 2.f;

	glm::vec3 pLens(0.f);
	glm::vec3 pFocusPlane = glm::vec3(ruv * glm::vec2(aspect, 1.f) * tanFovY, 1.f) * cam.focalDist;
	glm::vec3 dir = pFocusPlane - pLens;

	Ray ray;
	ray.direction = glm::normalize(glm::mat3(cam.right, cam.up, cam.view) * dir);
	ray.origin = cam.position + cam.right * pLens.x + cam.up * pLens.y;

	Intersection intersec;
	scene->intersect(ray, intersec);

	if (intersec.primId != NullPrimitive) {
		if (scene->materials[intersec.matId].type == Material::Type::Light) {
#if SCENE_LIGHT_SINGLE_SIDED
			if (glm::dot(intersec.norm, ray.direction) < 0.f) {
				intersec.primId = NullPrimitive;
			}
#endif
		}
		else {
			intersec.wo = -ray.direction;
		}
	}
	GBuffer[idy * cam.resolution.x + idx] = intersec;
}

void pathTraceInit(Scene* scene) {
	hstScene = scene;

	const Camera& cam = hstScene->camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&devImage, pixelcount * sizeof(glm::vec3));
	hipMemset(devImage, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&devPaths, pixelcount * sizeof(PathSegment));
	hipMalloc(&devTerminatedPaths, pixelcount * sizeof(PathSegment));
	devPathsThr = thrust::device_ptr<PathSegment>(devPaths);
	devTerminatedPathsThr = thrust::device_ptr<PathSegment>(devTerminatedPaths);

	hipMalloc(&devIntersections, pixelcount * sizeof(Intersection));
	hipMemset(devIntersections, 0, pixelcount * sizeof(Intersection));
	devIntersectionsThr = thrust::device_ptr<Intersection>(devIntersections);

	hipMalloc(&devIntersecMatKeys, pixelcount * sizeof(int));
	hipMalloc(&devSegmentMatKeys, pixelcount * sizeof(int));
	devIntersecMatKeysThr = thrust::device_ptr<int>(devIntersecMatKeys);
	devSegmentMatKeysThr = thrust::device_ptr<int>(devSegmentMatKeys);
	checkCUDAError("pathTraceInit");

#if ENABLE_GBUFFER
	hipMalloc(&devGBuffer, pixelcount * sizeof(Intersection));
	const int BlockSize = 8;
	dim3 blockSize(BlockSize, BlockSize);

	dim3 blockNum((cam.resolution.x + BlockSize - 1) / BlockSize,
		(cam.resolution.y + BlockSize - 1) / BlockSize
	);
	renderGBuffer<<<blockNum, blockSize>>>(hstScene->devScene, cam, devGBuffer);
	checkCUDAError("GBuffer");
	std::cout << "[GBuffer generated]" << std::endl;
#endif
}

void pathTraceFree() {
	cudaSafeFree(devImage);  // no-op if devImage is null
	cudaSafeFree(devPaths);
	cudaSafeFree(devTerminatedPaths);
	cudaSafeFree(devIntersections);
	cudaSafeFree(devIntersecMatKeys);
	cudaSafeFree(devSegmentMatKeys);
#if ENABLE_GBUFFER
	cudaSafeFree(devGBuffer);
#endif
}

/**
 * Antialiasing and physically based camera (lens effect)
 */
__device__ Ray sampleCamera(DevScene* scene, const Camera& cam, int x, int y, glm::vec4 r) {
	Ray ray;
#if CAMERA_PANORAMA
	float u = (x - .5f + r.x) / cam.resolution.x - .5f;
	float v = (y - .5f + r.y) / cam.resolution.y;
	glm::vec3 dir = Math::toSphere(glm::vec2(u, v));
	dir = cam.right * dir.x + cam.up * dir.y + cam.view * dir.z;
	ray.direction = dir;
	ray.origin = cam.position;
#else
	float aspect = float(cam.resolution.x) / cam.resolution.y;
	float tanFovY = glm::tan(glm::radians(cam.fov.y));
	glm::vec2 pixelSize = 1.f / glm::vec2(cam.resolution);
	glm::vec2 scr = glm::vec2(x, y) * pixelSize;
	glm::vec2 ruv = scr + pixelSize * glm::vec2(r.x, r.y);
	ruv = 1.f - ruv * 2.f;

	glm::vec2 pAperture;
	if (scene->apertureMask != nullptr) {
		int id = scene->apertureSampler.sample(r.z, r.w);
		pAperture.x = glm::fract((id + .5f) / scene->apertureMask->width);
		pAperture.y = (id / scene->apertureMask->width + .5f) / scene->apertureMask->height;
		pAperture = pAperture * 2.f - 1.f;
	}
	else {
		pAperture = Math::toConcentricDisk(r.z, r.w);
	}

	glm::vec3 pLens = glm::vec3(pAperture * cam.lensRadius, 0.f);

	glm::vec3 pFocusPlane = glm::vec3(ruv * glm::vec2(aspect, 1.f) * tanFovY, 1.f) * cam.focalDist;
	glm::vec3 dir = pFocusPlane - pLens;
	ray.direction = glm::normalize(glm::mat3(cam.right, cam.up, cam.view) * dir);
	ray.origin = cam.position + cam.right * pLens.x + cam.up * pLens.y;
#endif
	return ray;
}

__global__ void generateRayFromCamera(
	DevScene* scene, Camera cam, 
	int iter, int traceDepth, PathSegment* pathSegments
) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];
		Sampler rng = makeSeededRandomEngine(iter, index, traceDepth, scene->sampleSequence);

		segment.ray = sampleCamera(scene, cam, x, y, sample4D(rng));
		segment.throughput = glm::vec3(1.f);
		segment.radiance = glm::vec3(0.f);
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void previewGBuffer(int iter, DevScene* scene, Camera cam, glm::vec3* image, int kind) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= cam.resolution.x || y >= cam.resolution.y) {
		return;
	}
	int index = y * cam.resolution.x + x;
	Sampler rng = makeSeededRandomEngine(iter, index, 0, scene->sampleSequence);

	Ray ray = sampleCamera(scene, cam, x, y, sample4D(rng));
	Intersection intersec;
	scene->intersect(ray, intersec);

	if (kind == 0) {
		image[index] += intersec.pos;
	}
	else if (kind == 1) {
		if (intersec.primId != NullPrimitive) {
			Material m = scene->getTexturedMaterialAndSurface(intersec);
		}
		image[index] += (intersec.norm + 1.f) * .5f;
	}
	else if (kind == 2) {
		image[index] += glm::vec3(intersec.uv, 1.f);
	}
}

__global__ void computeIntersections(
	int depth,
	int numPaths,
	PathSegment* pathSegments,
	DevScene* scene,
	Intersection* intersections,
	int* materialKeys,
	bool sortMaterial
#if ENABLE_GBUFFER
	, Intersection* GBuffer
#endif
) {
	int pathIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pathIdx >= numPaths) {
		return;
	}

	Intersection intersec;
	PathSegment segment = pathSegments[pathIdx];
#if ENABLE_GBUFFER
	if (depth == 0) {
		intersections[pathIdx] = GBuffer[pathIdx];
		return;
	}
#endif

#if BVH_DISABLE
	scene->naiveIntersect(segment.ray, intersec);
#else
	scene->intersect(segment.ray, intersec);
#endif

	if (intersec.primId != NullPrimitive) {
		if (scene->materials[intersec.matId].type == Material::Type::Light) {
#if SCENE_LIGHT_SINGLE_SIDED
			if (glm::dot(intersec.norm, segment.ray.direction) < 0.f) {
				intersec.primId = NullPrimitive;
			}
			else
#endif
			if (depth != 0) {
				// If not first ray, preserve previous sampling information for
				// MIS calculation
				intersec.prevPos = segment.ray.origin;
				if (sortMaterial) {
					intersec.prev = segment.prev;
				}
			}
		}
		else {
			intersec.wo = -segment.ray.direction;
		}
		if (sortMaterial) {
			materialKeys[pathIdx] = intersec.matId;
		}
	}
	else if (sortMaterial) {
		materialKeys[pathIdx] = -1;
	}
	intersections[pathIdx] = intersec;
}

__global__ void pathIntegSampleSurface(
	int iter,
	int depth,
	PathSegment* segments,
	Intersection* intersections,
	DevScene* scene,
	int numPaths,
	bool sortMaterial
) {
	const int SamplesOneIter = 7;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= numPaths) {
		return;
	}
	Intersection intersec = intersections[idx];
	PathSegment& segment = segments[idx];

	if (intersec.primId == NullPrimitive) {
		if (scene->envMap != nullptr) {
			if (scene->envMap != nullptr) {
				glm::vec3 w = segment.ray.direction;
				glm::vec3 radiance = scene->envMap->linearSample(Math::toPlane(w)) * segment.throughput;

				if (depth == 0) {
					segment.radiance += radiance * segment.throughput;
				}
				else {
					float weight = segment.prev.deltaSample ? 1.f :
						Math::powerHeuristic(segment.prev.BSDFPdf, scene->environmentMapPdf(w));
					segment.radiance += radiance * weight;
				}
			}
		}
		segment.remainingBounces = 0;

		if (Math::luminance(segment.radiance) < 1e-4f) {
			segment.pixelIndex = PixelIdxForTerminated;
		}
		return;
	}

	Sampler rng = makeSeededRandomEngine(iter, idx, 4 + depth * SamplesOneIter, scene->sampleSequence);

	Material material = scene->getTexturedMaterialAndSurface(intersec);

	glm::vec3 accRadiance(0.f);

	if (material.type == Material::Type::Light) {
		PrevBSDFSampleInfo prev = sortMaterial ? intersec.prev : segment.prev;

		glm::vec3 radiance = material.baseColor;
		if (depth == 0) {
			accRadiance += radiance;
		}
		else if (prev.deltaSample) {
			accRadiance += radiance * segment.throughput;
		}
		else {
			float lightPdf = Math::pdfAreaToSolidAngle(Math::luminance(radiance) * scene->sumLightPowerInv,
				intersec.prevPos, intersec.pos, intersec.norm);
			float BSDFPdf = prev.BSDFPdf;
			accRadiance += radiance * segment.throughput * Math::powerHeuristic(BSDFPdf, lightPdf);
		}
		segment.remainingBounces = 0;
	}
	else {
		bool deltaBSDF = (material.type == Material::Type::Dielectric);
		if (material.type != Material::Type::Dielectric && glm::dot(intersec.norm, intersec.wo) < 0.f) {
			intersec.norm = -intersec.norm;
		}

		if (!deltaBSDF) {
			glm::vec3 radiance;
			glm::vec3 wi;
			float lightPdf = scene->sampleDirectLight(intersec.pos, sample4D(rng), radiance, wi);

			if (lightPdf > 0.f) {
				float BSDFPdf = material.pdf(intersec.norm, intersec.wo, wi);
				accRadiance += segment.throughput * material.BSDF(intersec.norm, intersec.wo, wi) *
					radiance * Math::satDot(intersec.norm, wi) / lightPdf * Math::powerHeuristic(lightPdf, BSDFPdf);
			}
		}

		BSDFSample sample;
		material.sample(intersec.norm, intersec.wo, sample3D(rng), sample);

		if (sample.type == BSDFSampleType::Invalid) {
			// Terminate path if sampling fails
			segment.remainingBounces = 0;
		}
		else if (sample.pdf < 1e-8f) {
			segment.remainingBounces = 0;
		}
		else {
			bool deltaSample = (sample.type & BSDFSampleType::Specular);
			segment.throughput *= sample.bsdf / sample.pdf *
				(deltaSample ? 1.f : Math::absDot(intersec.norm, sample.dir));
			segment.ray = makeOffsetedRay(intersec.pos, sample.dir);
			segment.prev = { sample.pdf, deltaSample };
			segment.remainingBounces--;
		}
	}
	segment.radiance += accRadiance;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths) {
		PathSegment iterationPath = iterationPaths[index];
		if (iterationPath.pixelIndex >= 0 && iterationPath.remainingBounces <= 0) {
			glm::vec3 r = iterationPath.radiance;
			if (isnan(r.x) || isnan(r.y) || isnan(r.z) || isinf(r.x) || isinf(r.y) || isinf(r.z)) {
				return;
			}
			image[iterationPath.pixelIndex] += glm::clamp(r, glm::vec3(0.f), glm::vec3(FLT_MAX / 10.f));
		}
	}
}

__global__ void singleKernelPT(int iter, int maxDepth, DevScene* scene, Camera cam, glm::vec3* image) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= cam.resolution.x || y >= cam.resolution.y) {
		return;
	}
	glm::vec3 accRadiance(0.f);

	int index = y * cam.resolution.x + x;
	Sampler rng = makeSeededRandomEngine(iter, index, 0, scene->sampleSequence);

	Ray ray = sampleCamera(scene, cam, x, y, sample4D(rng));
	Intersection intersec;
	scene->intersect(ray, intersec);

	if (intersec.primId == NullPrimitive) {
		if (scene->envMap != nullptr) {
			glm::vec2 uv = Math::toPlane(ray.direction);
			accRadiance += scene->envMap->linearSample(uv);
		}
		goto WriteRadiance;
	}

	Material material = scene->getTexturedMaterialAndSurface(intersec);

	if (material.type == Material::Type::Light) {
		if (glm::dot(intersec.norm, ray.direction) > 0.f) {
			accRadiance = material.baseColor;
		}
		goto WriteRadiance;
	}

	glm::vec3 throughput(1.f);
	intersec.wo = -ray.direction;

	for (int depth = 1; depth <= maxDepth; depth++) {
		bool deltaBSDF = (material.type == Material::Type::Dielectric);

		if (material.type != Material::Type::Dielectric && glm::dot(intersec.norm, intersec.wo) < 0.f) {
			intersec.norm = -intersec.norm;
		}

		if (!deltaBSDF) {
			glm::vec3 radiance;
			glm::vec3 wi;
			float lightPdf = scene->sampleDirectLight(intersec.pos, sample4D(rng), radiance, wi);

			if (lightPdf > 0.f) {
				float BSDFPdf = material.pdf(intersec.norm, intersec.wo, wi);
				accRadiance += throughput * material.BSDF(intersec.norm, intersec.wo, wi) *
					radiance * Math::satDot(intersec.norm, wi) / lightPdf * Math::powerHeuristic(lightPdf, BSDFPdf);
			}
		}

		BSDFSample sample;
		material.sample(intersec.norm, intersec.wo, sample3D(rng), sample);

		if (sample.type == BSDFSampleType::Invalid) {
			// Terminate path if sampling fails
			break;
		}
		else if (sample.pdf < 1e-8f) {
			break;
		}

		bool deltaSample = (sample.type & BSDFSampleType::Specular);
		throughput *= sample.bsdf / sample.pdf *
			(deltaSample ? 1.f : Math::absDot(intersec.norm, sample.dir));
		
		ray = makeOffsetedRay(intersec.pos, sample.dir);

		glm::vec3 curPos = intersec.pos;
		scene->intersect(ray, intersec);
		intersec.wo = -ray.direction;

		if (intersec.primId == NullPrimitive) {
			if (scene->envMap != nullptr) {
				glm::vec3 radiance = scene->envMap->linearSample(Math::toPlane(ray.direction))
					* throughput;

				float weight = deltaSample ? 1.f :
					Math::powerHeuristic(sample.pdf, scene->environmentMapPdf(ray.direction));

				accRadiance += radiance * weight;
			}
			break;
		}
		material = scene->getTexturedMaterialAndSurface(intersec);

		if (material.type == Material::Type::Light) {
#if SCENE_LIGHT_SINGLE_SIDED
			if (glm::dot(intersec.norm, ray.direction) < 0.f) {
				break;
			}
#endif
			glm::vec3 radiance = material.baseColor;

			float weight = deltaSample ? 1.f : Math::powerHeuristic(
				sample.pdf,
				Math::pdfAreaToSolidAngle(Math::luminance(radiance) * scene->sumLightPowerInv,
					curPos, intersec.pos, intersec.norm)
			);
			accRadiance += radiance * throughput * weight;
			break;
		}
	}
WriteRadiance:
	if (isnan(accRadiance.x) || isnan(accRadiance.y) || isnan(accRadiance.z) ||
		isinf(accRadiance.x) || isinf(accRadiance.y) || isinf(accRadiance.z)) {
		return;
	}
	image[index] += accRadiance;
}

__global__ void BVHVisualize(int iter, DevScene* scene, Camera cam, glm::vec3* image) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= cam.resolution.x || y >= cam.resolution.y) {
		return;
	}
	int index = y * cam.resolution.x + x;

	Sampler rng = makeSeededRandomEngine(iter, index, 0, scene->sampleSequence);
	Ray ray = sampleCamera(scene, cam, x, y, sample4D(rng));

	Intersection intersec;
	scene->visualizedIntersect(ray, intersec);

	float logDepth = 0.f;
	int size = scene->BVHSize;
	while (size) {
		logDepth += 1.f;
		size >>= 1;
	}
	image[index] += glm::vec3(float(intersec.primId) / logDepth * .06f);
}

struct CompactTerminatedPaths {
	__host__ __device__ bool operator() (const PathSegment& segment) {
		return !(segment.pixelIndex >= 0 && segment.remainingBounces <= 0);
	}
};

struct RemoveInvalidPaths {
	__host__ __device__ bool operator() (const PathSegment& segment) {
		return segment.pixelIndex < 0 || segment.remainingBounces <= 0;
	}
};

void pathTrace(uchar4* pbo, int frame, int iter) {
	const Camera& cam = hstScene->camera;
	const int pixelCount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2D(8, 8);
	const dim3 blocksPerGrid2D(
		(cam.resolution.x + blockSize2D.x - 1) / blockSize2D.x,
		(cam.resolution.y + blockSize2D.y - 1) / blockSize2D.y);

	int depth = 0;
	int numPaths = pixelCount;

	auto devTerminatedThr = devTerminatedPathsThr;

	if (Settings::tracer == Tracer::Streamed) {
		generateRayFromCamera<<<blocksPerGrid2D, blockSize2D>>>(hstScene->devScene, cam, iter, Settings::traceDepth, devPaths);
		checkCUDAError("PT::generateRayFromCamera");
		hipDeviceSynchronize();

		bool iterationComplete = false;
		while (!iterationComplete) {
			// clean shading chunks
			hipMemset(devIntersections, 0, pixelCount * sizeof(Intersection));

			// tracing
			const int BlockSizeIntersec = 128;
			int blockNumIntersec = (numPaths + BlockSizeIntersec - 1) / BlockSizeIntersec;
			computeIntersections<<<blockNumIntersec, BlockSizeIntersec>>>(
				depth, numPaths, devPaths, hstScene->devScene, devIntersections, devIntersecMatKeys, Settings::sortMaterial
#if ENABLE_GBUFFER
				, devGBuffer
#endif
			);
			checkCUDAError("PT::computeInteractions");
			hipDeviceSynchronize();

			if (Settings::sortMaterial) {
				cudaMemcpyDevToDev(devSegmentMatKeys, devIntersecMatKeys, numPaths * sizeof(int));
				thrust::sort_by_key(devIntersecMatKeysThr, devIntersecMatKeysThr + numPaths, devIntersectionsThr);
				thrust::sort_by_key(devSegmentMatKeysThr, devSegmentMatKeysThr + numPaths, devPathsThr);
			}

			const int BlockSizeSample = 64;
			int blockNumSample = (numPaths + BlockSizeSample - 1) / BlockSizeSample;

			pathIntegSampleSurface<<<blockNumSample, BlockSizeSample>>>(
				iter, depth, devPaths, devIntersections, hstScene->devScene, numPaths, Settings::sortMaterial
			);
			checkCUDAError("PT::sampleSurface");
			hipDeviceSynchronize();

			// Compact paths that are terminated but carry contribution into a separate buffer
			devTerminatedThr = thrust::remove_copy_if(devPathsThr, devPathsThr + numPaths, devTerminatedThr, CompactTerminatedPaths());
			// Only keep active paths
			auto end = thrust::remove_if(devPathsThr, devPathsThr + numPaths, RemoveInvalidPaths());
			numPaths = end - devPathsThr;
			//std::cout << "Remaining paths: " << numPaths << "\n";

			iterationComplete = (numPaths == 0);
			depth++;

			if (guiData != nullptr) {
				guiData->TracedDepth = depth;
			}
		}

		// Assemble this iteration and apply it to the image
		const int BlockSizeGather = 128;
		dim3 numBlocksPixels = (pixelCount + BlockSizeGather - 1) / BlockSizeGather;
		int numContributing = devTerminatedThr.get() - devTerminatedPaths;
		finalGather<<<numBlocksPixels, BlockSizeGather>>>(numContributing, devImage, devTerminatedPaths);
	}
	else {
		const int BlockSizeSinglePTX = 8;
		const int BlockSizeSinglePTY = 8;
		int blockNumSinglePTX = (cam.resolution.x + BlockSizeSinglePTX - 1) / BlockSizeSinglePTX;
		int blockNumSinglePTY = (cam.resolution.y + BlockSizeSinglePTY - 1) / BlockSizeSinglePTY;

		dim3 singlePTBlockNum(blockNumSinglePTX, blockNumSinglePTY);
		dim3 singlePTBlockSize(BlockSizeSinglePTX, BlockSizeSinglePTY);

		if (Settings::tracer == Tracer::SingleKernel) {
			singleKernelPT<<<singlePTBlockNum, singlePTBlockSize>>>(iter, Settings::traceDepth, hstScene->devScene, cam, devImage);
		}
		else if (Settings::tracer == Tracer::BVHVisualize) {
			BVHVisualize<<<singlePTBlockNum, singlePTBlockSize>>>(iter, hstScene->devScene, cam, devImage);
		}
		else {
			previewGBuffer<<<singlePTBlockNum, singlePTBlockSize>>>(iter, hstScene->devScene, cam, devImage,
				Settings::GBufferPreviewOpt);
		}

		if (guiData != nullptr) {
			guiData->TracedDepth = Settings::traceDepth;
		}
	}

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2D, blockSize2D>>>(pbo, cam.resolution, iter, devImage, Settings::toneMapping);

	// Retrieve image from GPU
	hipMemcpy(hstScene->state.image.data(), devImage,
		pixelCount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathTrace");
}